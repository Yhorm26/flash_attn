#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <cmath>
#include <cstring>
#include <random>
#include <vector>
#include <algorithm>
#include <sys/time.h>
#include <hip/hip_fp16.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "utils.h"
#include "init_curand_states.h"
#include "flash_forward.h"
#include "flash_forward_splitkv.h"


#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

std::default_random_engine generator(26);
std::uniform_real_distribution<float> distribution(0.0f, 10.0f);

// 核函数指针类型定义
template<bool Is_dropout, bool Is_causal, bool Is_local, bool Has_alibi>
using KernelFunctionPtr = void (*)(mykernelParamType);

void verfiy(float* O, float* O_host, const int batch_size, const int n_heads, const int seq_len, const int head_dim, float range_of_error);

void attention_forward_cpu(float* Q, float* K, float* V, float softmax_scale, const int batch_size, const int n_heads, const int seq_len, 
    const int head_dim, float* output, const bool use_causal_mask = false, int window_size = -1, float* alibi_slopes = nullptr);

int num_splits_heuristic(int batch_nheads_mblocks, int num_SMs, int num_n_blocks, int max_splits, int seq_len);


int main(){
    const int  batch_size       = 1;
    const int  n_heads          = 8;
    const int  seq_len          = 1024;
    const int  head_dim         = 32;

    const bool dropout          = false;      // 一旦启用dropout,那核函数的结果和没有使用dropout的cpu端结果必然不同,因此便不再验证结果正确性
    const bool causal_mask      = true;     // 一般来说， causal_mask不会和window_attention同时启用 
    const bool window_attention = false;
    const bool alibi            = true;
    const bool Is_even_K        = !(head_dim % 32);
    const bool Is_even_MN       = !(seq_len % 128);

    float dropout_prob = 0.0f;
    hiprandStatePhilox4_32_10_t* d_states;
    if(dropout){
        dropout_prob = 0.1f;
    }

    int window_size = -1;
    if(window_attention){
        window_size = 128;
    }
    float *alibi_slopes = nullptr;
    float *alibi_slopes_device = nullptr;
    if(alibi){
        alibi_slopes = (float*)malloc(n_heads*sizeof(float));
        for (int i = 0; i < n_heads; i++){
            alibi_slopes[i] = -std::pow(2, -8.0 / n_heads * (i + 1));
        }
        hipMalloc((void**)&alibi_slopes_device, n_heads*sizeof(float));
        hipMemcpy(alibi_slopes_device, alibi_slopes, n_heads*sizeof(float),hipMemcpyHostToDevice);
    }
      
    float *Q      = (float*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(float));
    float *K      = (float*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(float));
    float *V      = (float*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(float));
    float *O      = (float*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(float));
    float *O_host = (float*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(float));

    half *Q_half = (half*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(half));
    half *K_half = (half*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(half));
    half *V_half = (half*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(half));

    float *O_device;
    half  *Q_device_half,*K_device_half,*V_device_half;
    hipMalloc((void**)&O_device, batch_size*n_heads*seq_len*head_dim*sizeof(float));
    hipMalloc((void**)&Q_device_half, batch_size*n_heads*seq_len*head_dim*sizeof(half));
    hipMalloc((void**)&K_device_half, batch_size*n_heads*seq_len*head_dim*sizeof(half));
    hipMalloc((void**)&V_device_half, batch_size*n_heads*seq_len*head_dim*sizeof(half));

    float* O_tmp; float* L; float* M;

    for(int i = 0; i < batch_size*n_heads*seq_len*head_dim; i++)
    {
        Q[i] = distribution(generator);
        K[i] = distribution(generator);
        V[i] = distribution(generator);
        O[i] = 0.0f;

        Q_half[i] = __float2half(Q[i]);
        K_half[i] = __float2half(K[i]);
        V_half[i] = __float2half(V[i]);
    }

    hipMemcpy(Q_device_half, Q_half, batch_size*n_heads*seq_len*head_dim*sizeof(half),hipMemcpyHostToDevice);
    hipMemcpy(K_device_half, K_half, batch_size*n_heads*seq_len*head_dim*sizeof(half),hipMemcpyHostToDevice);
    hipMemcpy(V_device_half, V_half, batch_size*n_heads*seq_len*head_dim*sizeof(half),hipMemcpyHostToDevice);
    
    mykernelParamType param;
    param.Q                 = Q_device_half;
    param.K                 = K_device_half;
    param.V                 = V_device_half;
    param.O                 = O_device;
    param.N                 = seq_len;
    param.d                 = head_dim;
    param.Br                = 128;
    param.Bc                = 128;
    param.Tc                = ceil(seq_len / param.Bc);
    param.Tr                = ceil(seq_len / param.Br);
    param.softmax_scale     = 1.0 / sqrt(head_dim);
    param.window_size_right = window_size;
    param.window_size_left  = window_size;
    param.alibi_slopes_ptr  = alibi_slopes_device;

    int split_num = num_splits_heuristic(param.Tr * n_heads * batch_size, 108, param.Tr, seq_len / 128, seq_len);

    if(dropout){
        // 分配状态内存
        int num_blocks = param.Tr * n_heads * batch_size * split_num * 256;
        hipMalloc(&d_states, num_blocks * sizeof(hiprandStatePhilox4_32_10_t));

        // 初始化状态
        dim3 grid((num_blocks + 255)/256, 1, 1);
        int seed = 48;
        init_curand_states<<<grid, 256>>>(d_states, seed, num_blocks);
        param.dropout_prob      = dropout_prob;
        param.states            = d_states;
    }

    // CPU端计算正确结果
    attention_forward_cpu(Q, K, V, param.softmax_scale, batch_size, n_heads, seq_len, head_dim, O, causal_mask, window_size, alibi_slopes);

    KernelFunctionPtr<dropout, causal_mask, window_attention, alibi> selectedKernel = nullptr;
    int griddim_x = 1, griddim_y = 1, griddim_z = 1;       // 初始化网格大小
    int blockdim_x = 1, blockdim_y = 1, blockdim_z = 1;    // 初始化线程块大小
    int sram_size;

    // 开始根据数据维度来选择核函数
    if(split_num == 1){
        // GPU网格尺寸
        griddim_x = param.Tr;
        griddim_y = n_heads;
        griddim_z = batch_size;
        // GPU线程块尺寸
        blockdim_x = 256;
        if(Is_even_K && Is_even_MN){
            // 共享内存大小
            sram_size = (param.Br + param.Bc * 2) * param.d * sizeof(half) + param.Br * param.d * sizeof(float);
            selectedKernel = forward_kernel<dropout, causal_mask, window_attention, alibi>;
        }
        else{
            int d_align32 = (param.d + 31) / 32 * 32;
            sram_size = (param.Br + param.Bc * 2) * d_align32 * sizeof(half) + param.Br * d_align32 * sizeof(float);
            // selectedKernel = forward_kernel_general<dropout, causal_mask, window_attention, alibi>;
        }
    }
    else{
        // GPU网格尺寸
        griddim_x = param.Tr * split_num;
        griddim_y = n_heads;
        griddim_z = batch_size; 
        // GPU线程块尺寸
        blockdim_x = 256;

        hipMalloc((void**)&O_tmp, batch_size*n_heads*seq_len*head_dim*split_num*sizeof(float));
        hipMalloc((void**)&L, batch_size*n_heads*seq_len*split_num*sizeof(float));
        hipMalloc((void**)&M, batch_size*n_heads*seq_len*split_num*sizeof(float));

        param.O_tmp = O_tmp;
        param.split_num = split_num;
        param.L = L;
        param.M = M;

        if(Is_even_K && Is_even_MN){
            // 共享内存大小
            sram_size = (param.Br + param.Bc * 2) * param.d * sizeof(half) + param.Br * param.d * sizeof(float);
            selectedKernel = forward_kernel_splitkv<dropout, causal_mask, window_attention, alibi>;
        }
        else{
            int d_align32 = (param.d + 31) / 32 * 32;
            sram_size = (param.Br + param.Bc * 2) * d_align32 * sizeof(half) + param.Br * d_align32 * sizeof(float);
            // selectedKernel = forward_kernel_splitkv_general<dropout, causal_mask, window_attention, alibi>;
        }
    }

    dim3 grid_dim(griddim_x, griddim_y, griddim_z);
    dim3 block_dim(blockdim_x, blockdim_y, blockdim_z);
    
    // 计时
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    float time_elapsed=0.0;
    // 核函数启动
    selectedKernel<<<grid_dim, block_dim, sram_size>>>(param);
    if(split_num > 1){
        dim3 combine_grid_dim(param.Tr, n_heads, batch_size);
        forward_kernel_splitkv_combine<<<combine_grid_dim, 128>>>(param);
    }    
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // 计时结束
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed,start,stop);

    // 将GPU结果拷贝回主机端
    hipMemcpy(O_host, O_device, batch_size*n_heads*seq_len*head_dim*sizeof(float), hipMemcpyDeviceToHost);
    printf("kernel time: %f us\n", time_elapsed*1000);
    // 检验结果正确性
    if(!dropout){
        printf("Verify the result of kernel function\n");
        verfiy(O, O_host, batch_size, n_heads, seq_len, head_dim, 0.05);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // 释放显存
    hipFree(O_device);
    hipFree(Q_device_half);
    hipFree(K_device_half);
    hipFree(V_device_half);
    hipFree(d_states);
    
    hipFree(L);
    hipFree(M);
    hipFree(O_tmp);
    
    // 释放内存
    free(Q);
    free(K);
    free(V);
    free(O);
    free(O_host);
    free(Q_half);
    free(K_half);
    free(V_half);
    
    return 0;
}


void verfiy(
    float* O, 
    float* O_host,
    const int batch_size,
    const int n_heads,
    const int seq_len,
    const int head_dim,
    float range_of_error)
{
    int error=0;
    printf("===================start verfiy===================\n");
    for(int i=0;i<batch_size*n_heads*seq_len*head_dim;i++)
    {
        float device_out = O_host[i];
        if((fabs(O_host[i] - O[i]))/O_host[i] > range_of_error || std::isnan(device_out) || std::isinf(device_out))
        {
            printf("error, postion:%d, gpuvalue:%f, cpuvalue:%f\n", i, O_host[i], O[i]);
            error++;
            break;
        }        
    }
    printf("==================finish,error:%d==================\n",error);
}


void attention_forward_cpu(
    float* Q, 
    float* K, 
    float* V, 
    float softmax_scale,
    const int batch_size,
    const int n_heads,
    const int seq_len, 
    const int head_dim, 
    float* output,
    const bool use_causal_mask,
    int window_size,
    float* alibi_slopes)
{
    const int head_size = seq_len * head_dim;
    const int seq_sq = seq_len * seq_len;

    // 临时存储注意力分数
    float* scores = new float[seq_sq];

    for (int b = 0; b < batch_size; ++b) {
        for (int h = 0; h < n_heads; ++h) {
            // 获取当前head的指针偏移量
            const int base_offset = b * n_heads * head_size + h * head_size;
            const float* Q_ptr = Q + base_offset;
            const float* K_ptr = K + base_offset;
            const float* V_ptr = V + base_offset;
            float* out_ptr = output + base_offset;

            // 1. 计算QK^T
            for (int i = 0; i < seq_len; ++i) {
                for (int j = 0; j < seq_len; ++j) {
                    float sum = 0.0f;
                    for (int k = 0; k < head_dim; ++k) {
                        sum += Q_ptr[i * head_dim + k] * K_ptr[j * head_dim + k];
                    }
                    scores[i * seq_len + j] = sum * softmax_scale;
                }
            }

            // 2. 应用ALiBi偏置
            if (alibi_slopes != nullptr) {
                const float slope = alibi_slopes[h];
                for (int i = 0; i < seq_len; ++i) {
                    for (int j = 0; j < seq_len; ++j) {
                        scores[i * seq_len + j] -= slope * std::abs(i - j);
                    }
                }
            }

            // 3. 应用注意力掩码
            if (use_causal_mask) {
                for (int i = 0; i < seq_len; ++i) {
                    for (int j = 0; j < seq_len; ++j) {
                        if (j > i) {
                            scores[i * seq_len + j] = -INFINITY;
                        }
                    }
                }
            }

            if (window_size >= 0) {
                const int w = window_size;
                for (int i = 0; i < seq_len; ++i) {
                    for (int j = 0; j < seq_len; ++j) {
                        if (std::abs(i - j) > w) {
                            scores[i * seq_len + j] = -INFINITY;
                        }
                    }
                }
            }

            // 4. Softmax计算
            for (int i = 0; i < seq_len; ++i) {
                float max_val = -INFINITY;
                float* row = scores + i * seq_len;
                
                // 计算行最大值
                for (int j = 0; j < seq_len; ++j) {
                    max_val = std::max(max_val, row[j]);
                }

                // 计算指数和
                float sum = 0.0f;
                for (int j = 0; j < seq_len; ++j) {
                    row[j] = expf(row[j] - max_val);
                    sum += row[j];
                }

                // 归一化
                for (int j = 0; j < seq_len; ++j) {
                    row[j] /= sum;
                }
            }

            // 5. 计算加权和
            for (int i = 0; i < seq_len; ++i) {
                for (int k = 0; k < head_dim; ++k) {
                    float sum = 0.0f;
                    for (int j = 0; j < seq_len; ++j) {
                        sum += scores[i * seq_len + j] * V_ptr[j * head_dim + k];
                    }
                    out_ptr[i * head_dim + k] = sum;
                }
            }
        }
    }

    delete[] scores;
}


int num_splits_heuristic(int batch_nheads_mblocks, int num_SMs, int num_n_blocks, int max_splits, int seq_len) {
    if (batch_nheads_mblocks >= 4.0f * num_SMs) { return 1; }
    max_splits = std::min({max_splits, num_SMs, num_n_blocks});
    float max_efficiency = 0.f;
    std::vector<float> efficiency;
    efficiency.reserve(max_splits);
    auto ceildiv = [](int a, int b) { return (a + b - 1) / b; };
    auto is_split_eligible = [&ceildiv, &num_n_blocks, &seq_len](int num_splits) {
        return (num_splits == 1 || ceildiv(num_n_blocks, num_splits) != ceildiv(num_n_blocks, num_splits - 1)) && (seq_len % (128 * num_splits) == 0);
    };
    for (int num_splits = 1; num_splits <= max_splits; num_splits++) {
        if (!is_split_eligible(num_splits)) {
            efficiency.push_back(0.f);
        } else {
            // 计算拆分后的平均efficiency
            float n_waves = float(batch_nheads_mblocks * num_splits) / num_SMs;
            float eff = n_waves / ceil(n_waves);
            // printf("num_splits = %d, eff = %f\n", num_splits, eff);
            if (eff > max_efficiency) { max_efficiency = eff; }
            efficiency.push_back(eff);
        }
    }
    // 选择满足85%利用率最小的拆分
    for (int num_splits = 1; num_splits <= max_splits; num_splits++) {
        if (!is_split_eligible(num_splits)) { continue; }
        if (efficiency[num_splits - 1] >= 0.85 * max_efficiency) {
            // printf("num_splits chosen = %d\n", num_splits);
            return num_splits;
        }
    }
    return 1;
}
